/*
 * transfer.cpp
 *
 *  Created on: 2012-9-5
 *      Author: Alexander
 */

#include "domain_service_algOutput_AlgGpuRunStep.h"
#include <jni.h>
#include <stdio.h>
#include <stdlib.h>
#include "jniTransform.h"
#include "cudaTool.cuh"
static void initContext(JNIEnv*,jobject);
static void cToJava(JNIEnv*,jobject);
static int populationSize;
static int rowNum;
static int columnNum;
static int normalGeneLength;
static int homeoticGeneLength;
static int homeoticGeneNum;
static int normalGeneNum;
static int selectionRange;
static float accuracy;
static float** dataSet;
static char** normalGeneType;
static char** homeoticGeneType;
static char** normalGeneIndex;
static char** homeoticGeneIndex;
JNIEXPORT void JNICALL Java_domain_service_algOutput_AlgGpuRunStep_calcOnCuda(JNIEnv *env, jobject me, jobject gepAlgRun){
//	hipSetDevice(0);
//	printf("a\n");
//	fflush(stdout);
	initContext(env,gepAlgRun);
//	printf("b\n");
//	fflush(stdout);
	initcpu(populationSize,rowNum);
//	printf("c\n");
//	fflush(stdout);
	initgpu(normalGeneLength,normalGeneNum,populationSize,columnNum,rowNum,homeoticGeneLength,homeoticGeneNum);
//	printf("d\n");
//	fflush(stdout);
	cputogpu(normalGeneLength,normalGeneNum,populationSize,columnNum,rowNum,homeoticGeneLength,homeoticGeneNum,dataSet,normalGeneType,normalGeneIndex,homeoticGeneType,homeoticGeneIndex);
//	printf("e\n");
//	fflush(stdout);
	callKernel(normalGeneNum,homeoticGeneNum,populationSize,rowNum,columnNum,normalGeneLength,homeoticGeneLength,selectionRange,accuracy);
//	printf("f\n");
//	fflush(stdout);
	gputocpu(populationSize,rowNum);
//	printf("g\n");
//	fflush(stdout);
	cToJava(env,gepAlgRun);
//	printf("h\n");
//	fflush(stdout);
	freecpuandgpu(populationSize,normalGeneType,normalGeneIndex,homeoticGeneType,homeoticGeneIndex,dataSet);
//	hipDeviceReset();
	return;
}
static void initContext(JNIEnv *env,jobject gepAlgRun){
	iniAllId(env);
	populationSize=getPopulationSize(env,gepAlgRun);
	rowNum=getRowNum(env,gepAlgRun);
	columnNum=getColumnNum(env,gepAlgRun);
	normalGeneLength=getNormalGeneLength(env,gepAlgRun);
	normalGeneNum=getNormalGeneNum(env,gepAlgRun);
	homeoticGeneLength=getHomeoticGeneLength(env,gepAlgRun);
	homeoticGeneNum=getHomeoticGeneNum(env,gepAlgRun);
	selectionRange=getSelectionRange(env,gepAlgRun);
	accuracy=getAccuracy(env,gepAlgRun);
//	printf("a\n");
//	fflush(stdout);
	dataSet=createDataSet(env,gepAlgRun,rowNum,columnNum);
//	printf("b\n");
//	fflush(stdout);
	normalGeneType=createNormalGeneType(env,gepAlgRun,populationSize,normalGeneLength,normalGeneNum);
//	printf("c\n");
//	fflush(stdout);
	homeoticGeneType=createHomeoticGeneType(env,gepAlgRun,populationSize,homeoticGeneLength,homeoticGeneNum);
//	printf("d\n");
//	fflush(stdout);
	normalGeneIndex=createNormalGeneIndex(env,gepAlgRun,populationSize,normalGeneLength,normalGeneNum);
//	printf("e\n");
//	fflush(stdout);
	homeoticGeneIndex=createHomeoticGeneIndex(env,gepAlgRun,populationSize,homeoticGeneLength,homeoticGeneNum);
//	printf("f\n");
//	fflush(stdout);
}
static void cToJava(JNIEnv *env,jobject gepAlgRun){
//	printf("Hi\n");
//	fflush(stdout);
	toJavaFitness(env,gepAlgRun,getFitnessArray());
	toJavaFittedValue(env,gepAlgRun,getFittedValueArray());
	toJavaHomeoticGeneIndex(env,gepAlgRun,getHomeoticArray());
}
