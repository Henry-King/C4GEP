/*
 * transfer.cpp
 *
 *  Created on: 2012-9-5
 *      Author: Alexander
 */

#include "domain_service_algOutput_AlgGpuRunStep.h"
#include <jni.h>
#include <stdio.h>
#include <stdlib.h>
#include "jniTransform.h"
#include "cudaTool.cuh"
static void initContext(JNIEnv*,jobject);
static void cToJava(JNIEnv*,jobject);
static int populationSize;
static int rowNum;
static int columnNum;
static int normalGeneLength;
static int homeoticGeneLength;
static int homeoticGeneNum;
static int normalGeneNum;
static int selectionRange;
static float** dataSet;
static char** normalGeneType;
static char** homeoticGeneType;
static char** normalGeneIndex;
static char** homeoticGeneIndex;
JNIEXPORT void JNICALL Java_domain_service_algOutput_AlgGpuRunStep_calcOnCuda(JNIEnv *env, jobject me, jobject gepAlgRun){
	initContext(env,gepAlgRun);
	initcpu(populationSize,rowNum);
	initgpu(normalGeneLength,normalGeneNum,populationSize,columnNum,rowNum,homeoticGeneLength,homeoticGeneNum);
	cputogpu(normalGeneLength,normalGeneNum,populationSize,columnNum,rowNum,homeoticGeneLength,homeoticGeneNum,dataSet,normalGeneType,normalGeneIndex,homeoticGeneType,homeoticGeneIndex);
	callKernel(normalGeneNum,homeoticGeneNum,populationSize,rowNum,columnNum,normalGeneLength,homeoticGeneLength,selectionRange);
	gputocpu(populationSize,rowNum);
	cToJava(env,gepAlgRun);
	freecpuandgpu(populationSize,normalGeneType,normalGeneIndex,homeoticGeneType,homeoticGeneIndex);
	return;
}
static void initContext(JNIEnv *env,jobject gepAlgRun){
	iniAllId(env);
	dataSet=createDataSet(env,gepAlgRun);
	normalGeneType=createNormalGeneType(env,gepAlgRun);
	homeoticGeneType=createHomeoticGeneType(env,gepAlgRun);
	normalGeneIndex=createNormalGeneIndex(env,gepAlgRun);
	homeoticGeneIndex=createHomeoticGeneIndex(env,gepAlgRun);
	populationSize=getPopulationSize(env,gepAlgRun);
	rowNum=getRowNum(env,gepAlgRun);
	columnNum=getColumnNum(env,gepAlgRun);
	normalGeneLength=getNormalGeneLength(env,gepAlgRun);
	normalGeneNum=getNormalGeneNum(env,gepAlgRun);
	homeoticGeneLength=getHomeoticGeneLength(env,gepAlgRun);
	homeoticGeneNum=getHomeoticGeneNum(env,gepAlgRun);
	selectionRange=getSelectionRange(env,gepAlgRun);
}
static void cToJava(JNIEnv *env,jobject gepAlgRun){
//	printf("Hi\n");
//	fflush(stdout);
	toJavaFitness(env,gepAlgRun,getFitnessArray());
	toJavaFittedValue(env,gepAlgRun,getFittedValueArray());
	toJavaHomeoticGeneIndex(env,gepAlgRun,getHomeoticArray());
}
